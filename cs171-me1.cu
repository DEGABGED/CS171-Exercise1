#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void printDevProp(hipDeviceProp_t devProp) {
    printf("%s\n", devProp.name);
    printf("Major revision number:         %d\n", devProp.major);
    printf("Minor revision number:         %d\n", devProp.minor);
    printf("Total global memory:           %u bytes\n", devProp.totalGlobalMem);
    printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
    printf("Total shared memory per block: %u\n",devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n", devProp.regsPerBlock);
    printf("Warp size:                     %d\n", devProp.warpSize);
    printf("Maximum memory pitch:          %u\n", devProp.memPitch);
    printf("Total constant memory:         %u\n", devProp.totalConstMem);
    printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
    printf("Maximum threads per dimension: %d,%d,%d\n", devProp.maxThreadsDim[0], devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
    return;
}

void printMatrix(float *A, int dim) {
    printf("[\n");
    for (int i=0; i<dim; i++) {
        printf("  [");
        for (int j=0; j<dim; j++) {
            printf("%.2f, ", A[i*dim + j]);
        }
        printf("]\n");
    }
    printf("]\n");
}

__global__
void kernel_1t1e(float *d_A, float *d_B, float *d_C, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_A[idx] = d_B[idx] + d_C[idx];
    }
}

__global__
void kernel_1t1r(float *d_A, float *d_B, float *d_C, int rows) {
    int i = 0;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j < rows) {
        for(i = 0;i<rows;i++){
            d_A[i*rows + j] = d_B[i*rows + j] + d_C[i*rows + j];
        }
    }
}

__global__
void kernel_1t1c(float *d_A, float *d_B, float *d_C, int rows) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = 0;

    if (i < rows) {
        for(j = 0;j<rows;j++){
            d_A[i*rows + j] = d_B[i*rows + j] + d_C[i*rows + j];
        }
    }
}

void hostFunction(float *A, float *B, float *C, int rows) {
    // Allocate device memory
    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, rows*rows*sizeof(float));
    hipMalloc(&d_B, rows*rows*sizeof(float));
    hipMalloc(&d_C, rows*rows*sizeof(float));

    // Copy values to device memory
    hipMemcpy(d_B, B, rows*rows*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, rows*rows*sizeof(float), hipMemcpyHostToDevice);

    // Call kernel function
    int size = rows*rows;
    int numBlocks = (int) (rows/1024) + 1;
    dim3 threadsPerBlock(1024,1);

    //kernel_1t1e<<<(int) (rows/1024) + 1, 1024>>>(d_A, d_B, d_C, size);
    kernel_1t1r<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, rows);
    //kernel_1t1c<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, rows);
    hipDeviceSynchronize();

    // Get return value
    hipMemcpy(A, d_A, rows*rows*sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    // Device Query first
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printDevProp(deviceProp);
    }

    // In my (Francis) local machine there is only one CUDA machine, so I'll hardcode that one here
    // Allocate memory
    const int rows = 16;
    const int cols = rows;
    float *A, *B, *C;
    A = (float*) malloc(sizeof(float) * rows * cols);
    B = (float*) malloc(sizeof(float) * rows * cols);
    C = (float*) malloc(sizeof(float) * rows * cols);

    // Generate the values
    for (int i=0; i<rows; i++) {
        for (int j=0; j<cols; j++) {
            B[i*rows + j] = (float) rand() / (float) (RAND_MAX / 100);
            C[i*rows + j] = (float) rand() / (float) (RAND_MAX / 100);
        }
    }

    // Call the host function
    hostFunction(A, B, C, rows);

    printf("A:\n");
    printMatrix(A, rows);
    printf("B:\n");
    printMatrix(B, rows);
    printf("C:\n");
    printMatrix(C, rows);

    // Free memory
    free(A);
    free(B);
    free(C);

    printf("Done!\n");
}