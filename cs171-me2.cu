#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>

#define TILE_WIDTH 16

void printDevProp(hipDeviceProp_t devProp) {
    // Source: https://stackoverflow.com/questions/32530604/how-can-i-get-number-of-cores-in-cuda-device
    printf("%s\n", devProp.name);
    printf("Major revision number:         %d\n", devProp.major);
    printf("Minor revision number:         %d\n", devProp.minor);
    printf("Total global memory:           %u bytes\n", devProp.totalGlobalMem);
    printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
    printf("Total shared memory per block: %u\n",devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n", devProp.regsPerBlock);
    printf("Warp size:                     %d\n", devProp.warpSize);
    printf("Maximum memory pitch:          %u\n", devProp.memPitch);
    printf("Total constant memory:         %u\n", devProp.totalConstMem);
    printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
    printf("Maximum threads per dimension: %d,%d,%d\n", devProp.maxThreadsDim[0], devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
    return;
}

void printMatrix(float *A, int rows, int cols) {
    printf("[\n");
    for (int i=0; i<rows; i++) {
        printf("  [");
        for (int j=0; j<cols; j++) {
            printf("%.2f, ", A[i*cols + j]);
        }
        printf("]\n");
    }
    printf("]\n");
}

void printMatrixFlat(float *A, int rows, int cols) {
    printf("[");
    for (int i=0; i<rows; i++) {
        for (int j=0; j<cols; j++) {
            printf("%.2f, ", A[i*cols + j]);
        }
    }
    printf("]\n");
}

void populateMatrix(float *A, int rows, int cols) {
    // Generate the values
    for (int i=0; i<rows; i++) {
        for (int j=0; j<cols; j++) {
            A[i*cols + j] = (float) rand() / (float) (RAND_MAX / 100);
        }
    }
}

__global__
void matmul_rec_glob(float *d_A, float *d_B, float *d_C, int n, int m, int k) {
    int rows = blockIdx.y * blockDim.y + threadIdx.y;
    int cols = blockIdx.x * blockDim.x + threadIdx.x;

    if ((rows < n) && (cols < m)) {
        float val = 0;
        for (int i = 0; i < k; i++) {
            val += d_B[rows*k + i] * d_C[i*m + cols];
        }
        d_A[rows*m + cols] = val;
    }
}

__global__
void matmul_rec_shar(float *d_A, float *d_B, float *d_C, int n, int m, int k) {

    __shared__ float B_shared[TILE_WIDTH][TILE_WIDTH];
    __shared__ float C_shared[TILE_WIDTH][TILE_WIDTH];

    int rows = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int cols = blockIdx.x * TILE_WIDTH + threadIdx.x;

    if ((rows < n) && (cols < m)) {
        float val = 0;
        for (int i = 0; i<k/TILE_WIDTH; ++i){
            B_shared[threadIdx.y][threadIdx.x] = d_B[rows*k + i*TILE_WIDTH + threadIdx.x];
            C_shared[threadIdx.y][threadIdx.x] = d_C[(i*TILE_WIDTH + threadIdx.y)*k + cols];
            __syncthreads();
            for (int j = 0; j < TILE_WIDTH; ++j) {
                val += B_shared[threadIdx.y][j] * C_shared[j][threadIdx.x];
            }
            d_A[rows*k + cols] = val;
        }
    }
}

double hostFunction(float *A, float *B, float *C, int n, int m, int k, int blockSize, int kernel_choice) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate device memory
    // A = B * C
    // A: n x m
    // B: n x k
    // C: k x m
    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, n*m*sizeof(float));
    hipMalloc(&d_B, n*k*sizeof(float));
    hipMalloc(&d_C, k*m*sizeof(float));

    // Copy values to device memory
    hipMemcpy(d_B, B, n*k*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, k*m*sizeof(float), hipMemcpyHostToDevice);

    // Call kernel function
    const int dimY = 16;
    const int dimX = 16;
    dim3 dimBlock(dimX, dimY, 1);
    dim3 dimGrid(ceil((float) n / dimX), ceil((float) m / dimY), 1);

    hipEventRecord(start);
    if (kernel_choice == 0) {
        matmul_rec_glob<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n, m, k);
    }
    else if (kernel_choice == 1) {
        matmul_rec_shar<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n, m, k);
    }
    hipEventRecord(stop);
    hipDeviceSynchronize();

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Get return value
    hipMemcpy(A, d_A, n*m*sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return milliseconds;
}

int main() {
    // Device Query first
    int deviceCount;
    int blockSize = 1024;
    hipGetDeviceCount(&deviceCount);
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printDevProp(deviceProp);
        blockSize = deviceProp.maxThreadsPerBlock;
    }

    // In my (Francis) local machine there is only one CUDA machine, so I'll hardcode that one here
    // Allocate memory
    // A = B * C
    // A: n x m
    // B: n x k
    // C: k x m
    const int n = 16;
    const int m = 16;
    const int k = 16;
    float *A, *B, *C;
    A = (float*) malloc(sizeof(float) * n * m);
    B = (float*) malloc(sizeof(float) * n * k);
    C = (float*) malloc(sizeof(float) * k * m);

    // Benchmarking
    // Source: https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
    int kernel = 0;
    int runs = 5;
    double time_spent = 0.0;
    double ave_time = 0.0;
    printf("\n");

    while (kernel < 2) {
        printf("#%d:\t", kernel);
        for (int run=0; run<runs; run++) {
            populateMatrix(B, n, k);
            populateMatrix(C, k, m);
            time_spent = hostFunction(A, B, C, n, m, k, blockSize, kernel);
            ave_time += time_spent;
            printf("%.4f\t", time_spent);
        }
        ave_time /= runs;
        printf("Ave: %.4f\n", ave_time);
        kernel++;
    }

    // Check matrices
    printMatrix(A, n, m);
    printMatrix(B, n, k);
    printMatrix(C, k, m);

    // printMatrixFlat(A, n, m);
    // printMatrixFlat(B, n, k);
    // printMatrixFlat(C, k, m);

    // Free memory
    free(A);
    free(B);
    free(C);

    printf("\nDone!\n");
}